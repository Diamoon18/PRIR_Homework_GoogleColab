#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include<stdlib.h>
#include<time.h>

__global__ void saxpy_cuda(int n, float a, float *x, float *y){
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = tid; i < n; i += stride) {
        y[i] = a * x[i] + y[i];
    }
}

int main(){
    int n = 1<<25;
    float *x, *y, alpha=2.0;

    hipMalloc(&x, n * sizeof(float));
    hipMalloc(&y, n * sizeof(float));

    clock_t start, stop; 
	  for(int i = 1; i <= 50; i++){
        start=clock();
      
        saxpy_cuda<<<32, 1024>>>(n, alpha, x, y);
        hipDeviceSynchronize();

        stop = clock(); 
        double d = (double)(stop - start) * 1000.0 / CLOCKS_PER_SEC;                                                   
        printf("%lf\n", d);
    }	
    
    hipFree(x);
    hipFree(y);
}
